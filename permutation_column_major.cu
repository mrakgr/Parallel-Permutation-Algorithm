#include "hip/hip_runtime.h"
#include <thrust\device_vector.h>
#include <iostream>
#include <algorithm>
#include <chrono>
#include <ctime>

#include "hip/hip_runtime.h"
#include ""

using namespace thrust;
using namespace std;

unsigned long factorial(unsigned long n)
{
	return (n == 1 || n == 0) ? 1 : factorial(n - 1) * n;
}

__device__ unsigned long inline get_element(unsigned long *per_m, int row, int col, int per_num){
	if (row < 0) return 0;
	unsigned long pos = row + col*per_num;
	return per_m[pos];
}

__device__ void inline set_element(unsigned long *per_m, int row, int col, int per_num, unsigned long value){
	unsigned long pos = row + col*per_num;
	per_m[pos] = value;
}

__device__ unsigned long inline get_key_of_zero(unsigned long a, unsigned long n)
{
	a = ~a;              // search for 1-bit instead of 0-bit
	for (; n > 0; n--) {
		a = a & (a - 1);   // clear least significant 1-bit 
	}
	return 1UL << (__ffs(a) - 1);
}

void print_matrix_map(device_vector<unsigned long> matrix_map, unsigned long limit, unsigned long factor){
	// Prints only the first 100 rows, so the screen does not overflow.
	for (unsigned long i = 0; i < std::min(limit*factor, limit*100UL); i += limit){
		thrust::copy(matrix_map.begin() + i, matrix_map.begin() + i + limit, ostream_iterator<unsigned long>(cout, " "));
		cout << endl;
	}
}

__global__ void createPermutationMatrix_kernel(unsigned long *per_m, unsigned long factor, unsigned long per_num, int row, unsigned long factor2){
	factor; // The number of elements in a single row.
	factor2; // The variable that controls the variation in position as the program scans along the rows.
	unsigned long factor3 = per_num-row;
	per_m;  // A map of keys to the indices.

	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (col < factor){
		unsigned long m_previous, pos;
		m_previous = get_element(per_m, row - 1, col, per_num); // Get the element of the previous row.
		pos = col / factor2; // In the first row pos will range from (1...limit) in the next it range from (1...limit-1) and contract after that.
		pos = (pos % factor3) + 1;
		unsigned long idx = get_key_of_zero(m_previous, pos); // idx has only a single bit set to 1 at a location differing from that of m.
		unsigned long new_m = m_previous | idx; // That bit gets added here to the value of min the previous row.
		set_element(per_m, row, col, per_num, new_m); // matrix_map(row,column) is set to new_m.
	}
}

// What is happening here is similar to the optimization technique of dynamic programming where
// first a table is computed and then it is backtracked along the rows to compute the actual values.

// After the loop above, the all the elements of the first row should have only one bit set to 1 and all the elements of the last
// should have all the bits (up to the limit) set to 1. The second should have two bits set to one and the third should have
// three bits set to one and so on.

// To compute the key in the latest row that is being computed, you take the value of m in the previous row, take the bitwise
// (not arithmetic) negative of that and then apply m_previous BITWISE-AND m_current to compute the key. Then to get the index
// value from that you take the log_2 or simply the bitscan of that value to obtain the index.

__global__ void computeIndices_kernel(unsigned long *per_m, unsigned long factor, unsigned long per_num, int row){
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (col < factor){
		unsigned long m_previous, m_current;

		m_previous = get_element(per_m, row - 1, col, per_num);
		m_current = get_element(per_m, row, col, per_num);
		unsigned long new_m = (~m_previous) & m_current; // new_m should have only one bit set to 1.
		unsigned long index_of_new_m = __ffs(new_m) - 1;
		set_element(per_m, row, col, per_num, index_of_new_m);
	}
}

void createPermutationMatrix(unsigned long *d_perm, unsigned long perm_num)
{
	const unsigned long block_size = 512;

	unsigned long factor = factorial(perm_num);

	unsigned long gridx = factor / block_size;
	if (factor % block_size != 0) gridx++;

	unsigned long factor2 = factor / perm_num;
	for (int i = 0; i < perm_num; i++){
		createPermutationMatrix_kernel << <gridx, block_size >> >(d_perm, factor, perm_num, i, factor2);
		if (perm_num - i - 1 != 0) {
			factor2 /= (perm_num - i - 1);
		}
		hipDeviceSynchronize();
	}

	for (int i = perm_num-1; i >= 0; i--){
		computeIndices_kernel << <gridx, block_size >> >(d_perm, factor, perm_num, i);
		hipDeviceSynchronize();
	}



}

int main(){
	unsigned long num = 10; // Number of permutations.
	device_vector<unsigned long> perm(factorial(num)*num);

	unsigned long *d_perm = raw_pointer_cast(perm.data());

	std::chrono::time_point<std::chrono::high_resolution_clock> start, end;
	start = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < 1000; i++)
		createPermutationMatrix(d_perm, num);
	end = std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> elapsed_seconds = end - start;
	std::time_t end_time = std::chrono::high_resolution_clock::to_time_t(end);

	std::cout << "finished computation at " << std::ctime(&end_time)
		<< "elapsed time: " << elapsed_seconds.count() << "s\n";

	// On my GTX 970 a thousand iterations of generating 10! matrices gives results at 34.7 seconds. For comparison, the STL iterator would 
	// need 1.7 seconds on my OC'd i-4690k to compute a single matrix. The GPU version is roughly 50 times faster at generating permutation 
	// matrices.
	
	// The column major version of the algorithm as it does not have coalesced memory accesses is much slower than the row major version.

	print_matrix_map(perm, num, factorial(num));

	return 0;
}
